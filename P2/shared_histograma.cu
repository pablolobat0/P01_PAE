#include "hip/hip_runtime.h"
// General utilities
#include <stdlib.h>
#include <stdio.h>
// Math functions
#include <math.h>

// The file loadPGM.h will be used for defining load and export functions
#include "../pgmio.h"

#define SIZE 256
#define BLOCK_SIZE 1024

__global__ void histograma(unsigned char * d_xu8, int* v_hist, int dim){
    __shared__ int histo_s[SIZE];

    int tid = threadIdx.x;
    if(tid<SIZE)histo_s[tid]=0;
    __syncthreads();
    int i = threadIdx.x + blockIdx.x*blockDim.x;

    while (i<dim){ //loop por si el numero de hilos es menor que numero de pixeles
        atomicAdd(&histo_s[d_xu8[i]],1); //usamos atomic para evitar carreras criticas
        i+= blockIdx.x*blockDim.x;
    }
    __syncthreads();

    // Un solo hilo por nivel de intensidad actualiza la memoria global
    if (tid < SIZE) {
        atomicAdd(&v_hist[tid], histo_s[tid]);
    }
}


int main(int argc, char *argv[])
{ 
    // check for arguments
    if (argc < 2) {
    	printf("Use %s file.pgm\n", argv[0]);
        exit(-1);
    }

    // image width x height
    int w, h;

    // Load pgm image
    unsigned char* h_xu8 = loadPGMu8(argv[1], &w, &h);

    int h_hist[SIZE]= {0};

    unsigned char* v_xu8;
    int* v_hist;

    //reservamos memoria para las variables en memoria GLOBAL
    hipMalloc(&v_xu8,w*h*sizeof(unsigned char));
    hipMalloc(&v_hist, SIZE*sizeof(int));

    //inicalizamos variables en device
    hipMemset(v_hist,0,SIZE*sizeof(int));
    hipMemcpy(v_xu8,h_xu8,w*h*sizeof(unsigned char),hipMemcpyHostToDevice);
  
    int blockDim,blockCount;
    blockDim = BLOCK_SIZE;
    blockCount = (w * h + blockDim - 1) / blockDim;
    histograma<<<blockCount,blockDim>>>(v_xu8, v_hist, w*h);
    hipDeviceSynchronize();
     
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(err));
    }


    hipMemcpy(h_hist,v_hist,SIZE*sizeof(int),hipMemcpyDeviceToHost);

    free(h_xu8);
    hipFree(v_xu8);
    hipFree(v_hist);

    
    return 0;

}
