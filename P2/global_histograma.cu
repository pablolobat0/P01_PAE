#include "hip/hip_runtime.h"
// General utilities
#include <stdio.h>
#include <stdlib.h>
// Math functions
#include <math.h>
#include <sys/time.h>

// The file loadPGM.h will be used for defining load and export functions
#include "../pgmio.h"

#define SIZE 256
#define BLOCK_SIZE 256

__global__ void histograma(unsigned char *d_xu8, int *v_hist, int dim) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    // hay que poner un mutex, si no va a haber carreras criticas
    for (int i = idx; i < dim; i += stride) {
        atomicAdd(&v_hist[d_xu8[i]], 1);
    }
}

int main(int argc, char *argv[]) {
    int blockDim, blockCount;
    int width, height;
    int h_hist[SIZE] = {0};
    unsigned char *h_xu8, *v_xu8;
    int *valores_hist;
    struct timeval ex_start, ex_finish, init_start, init_finish;
    double time = 0;
    // check for arguments
    if (argc < 2) {
        printf("Use %s file.pgm\n", argv[0]);
        exit(-1);
    }

    gettimeofday(&init_start, NULL);
    // Load pgm image
    h_xu8 = loadPGMu8(argv[1], &width, &height);

    // reservamos memoria para las variables en memoria GLOBAL
    hipMalloc(&v_xu8, width * height * sizeof(unsigned char));
    hipMalloc(&valores_hist, SIZE * sizeof(int));

    // inicalizamos variables en device
    hipMemset(valores_hist, 0, SIZE * sizeof(int));
    hipMemcpy(v_xu8, h_xu8, width * height * sizeof(unsigned char),
               hipMemcpyHostToDevice);

    blockDim = BLOCK_SIZE;
    blockCount = (width * height + blockDim - 1) / blockDim;

    gettimeofday(&init_finish, NULL);



    gettimeofday(&ex_start, NULL);
    histograma<<<blockCount, blockDim>>>(v_xu8, valores_hist, width * height);
    hipDeviceSynchronize();
    gettimeofday(&ex_finish, NULL);

    hipMemcpy(h_hist, valores_hist, SIZE * sizeof(int),
               hipMemcpyDeviceToHost);

    for (int i = 0; i < SIZE; i++) {
        printf("%i ", h_hist[i]);
    }
    printf("\n");

    printf("NumBlocks=%d ------ BlockSize=%d\n", blockCount, blockDim);
    
    time = (init_finish.tv_sec - init_start.tv_sec +
            (init_finish.tv_usec - init_start.tv_usec) / 1.e6);

    printf("Reserva de memoria: %.10lf\n", time);

    time = (ex_finish.tv_sec - ex_start.tv_sec +
            (ex_finish.tv_usec - ex_start.tv_usec) / 1.e6);

    printf("Tiempo de Ejecucion: %.10lf\n", time);

    free(h_xu8);
    hipFree(v_xu8);
    hipFree(valores_hist);

    return 0;
}
